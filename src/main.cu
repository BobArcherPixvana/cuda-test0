#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hiprtc.h"

#include <chrono>
#include <cstddef>
#include <cstdint>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

void deviceQuery();

const char* logFile("log.txt");

#define LOG(exp)                                                                                   \
    do                                                                                             \
    {                                                                                              \
        std::ofstream ostr(logFile, std::ios::app);                                                \
        ostr << exp;                                                                               \
        std::cout << exp;                                                                          \
    } while(false)

#define INSPECT(exp) LOG(#exp << ": " << (exp) << "\n")
#define WHERE LOG(__FILE__ << ", " << __LINE__ << "\n")

namespace
{
std::string format(size_t i)
{
    std::ostringstream ostr;
    ostr << i;

    std::string s(ostr.str().c_str());
    std::string result;

    for(int i = 0; i < int(s.size()); ++i)
    {
        int const j(int(s.size()) - i - 1);

        if(i != 0 && i % 3 == 0)
        {
            result += ",";
        }

        result += s[j];
    }

    std::reverse(result.begin(), result.end());

    return result;
}

void writeResult(hipError_t result, std::string const& description)
{
    std::cout << std::setw(25) << std::left << description << " : ";
    std::cout << result << " ";
    std::cout << hipGetErrorName(result) << " ";
    std::cout << hipGetErrorString(result) << "\n";

    if(result != 0)
    {
        throw 1;
    }
}
}

__global__ void testKernel(int input, bool flip, int* pOutput)
{
    int const i(blockIdx.x * blockDim.x + threadIdx.x);
    printf("a");

    if(i == 0)
    {
        printf("b");
        *pOutput = flip ? 1 - input : input;
    }
}

int main(int argc, char* argv[])
{
    std::cout << "cuda-test0\n\n";

    std::chrono::high_resolution_clock::time_point startTime(
        std::chrono::high_resolution_clock::now());

    try
    {
        std::ofstream ostr(logFile, std::ios::trunc);
        ostr.close();

        size_t nBytes(1000000000);

        if(argc > 1)
        {
            nBytes = atol(argv[1]);
        }

        std::cout << format(nBytes) << " bytes\n\n";

        LOG(format(nBytes) << " bytes\n\n");

        unsigned char* src;
        unsigned char* dest;

        hipError_t rc(hipSuccess);

        // Allocate Unified Memory � accessible from CPU or GPU
        rc = hipMallocManaged(&src, nBytes);
        writeResult(rc, "hipMallocManaged");

        rc = hipMallocManaged(&dest, nBytes);
        writeResult(rc, "hipMallocManaged");
        /*
        // initialize x and y arrays on the host
        for(size_t i(0); i < nBytes; ++i)
        {
            src[i] = i % 256;
            dest[i] = 0;
        }
*/
        int const numBlocks(1);
        int const numThreadsPerThreadBlock(10);

        int* pInput;
        hipMallocManaged(&pInput, sizeof(int));

        int* pOutput;
        hipMallocManaged(&pOutput, sizeof(int));

        bool* pFlip;
        hipMallocManaged(&pFlip, sizeof(bool));

        *pInput = 17;
        *pOutput = -1000000;

        *pFlip = false;
        testKernel<<<numBlocks, numThreadsPerThreadBlock>>>(*pInput, *pFlip, pOutput);
        LOG("Start synchronize\n");
        hipDeviceSynchronize();
        LOG("Finish synchronize\n");

        INSPECT(*pInput);
        INSPECT(*pOutput);

        *pFlip = true;
        testKernel<<<numBlocks, numThreadsPerThreadBlock>>>(*pInput, *pFlip, pOutput);
        LOG("Start synchronize\n");
        hipDeviceSynchronize();
        LOG("Finish synchronize\n");

        INSPECT(*pInput);
        INSPECT(*pOutput);
    }
    catch(...)
    {
        std::cout << "Caught exception\n";
    }

    std::cout << std::fixed << std::setprecision(9);
    std::cout << std::chrono::duration_cast<std::chrono::duration<double>>(
                     std::chrono::high_resolution_clock::now() - startTime)
                     .count();

    return 0;
}
